#include "hip/hip_runtime.h"
/* Simple ground plane extraction operation.
 *
 * Ground plane extraction operations.
 */

// STL
#include <iostream>

// OpenVDB
#include <openvdb/math/Coord.h>

// NanoVDB
#include <nanovdb/NanoVDB.h>
#include <nanovdb/util/CudaDeviceBuffer.h>
#include <nanovdb/util/GridHandle.h>

// OVM
#include <openvdb_voxel_mapper/types.h>

namespace ovm::ops
{

// kernel to iterate through a single column of voxels and return the lowest point height
__global__ void min_z_kernel(const nanovdb::NanoGrid<uint32_t>& grid,
                             float* deviceMap,
                             const int xmin, const int xmax,
                             const int ymin, const int ymax,
                             const int zmin, const int zmax)
{
  // get the indices of this voxel (which correspond to the 2D grid, and the CUDA thread)
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;

  // ignore out of bounds threads
  if (i > xmax - xmin || j > ymax - ymin)
    return;

  // construct coordinate accessor for openvdb voxel in index space ([0,inf) -> (-inf, inf))
  const nanovdb::Coord ijk_begin {i + xmin, j + ymin, 0};

  // construct accessor and intra-voxel point iterators
  const nanovdb::Vec3f *begin = nullptr, *end = nullptr;

  // get device grid accessor
  auto acc = grid.getAccessor();

  // initialize minimum Z height
  float min_z = NAN;

  // starting from the first voxel, iterate upwards
  for (int k = zmin; k != zmax; ++k)
  {
    // get coordinate of this voxel
    const auto ijk = ijk_begin.offsetBy(0,0,k);
    const uint32_t offset = nanovdb::NanoLeaf<uint32_t>::CoordToOffset(ijk);

    // get leaf containing this voxel (and skip if inactive)
    auto* leaf = acc.probeLeaf(ijk);
    if (leaf == nullptr || !leaf->isActive(offset))
      continue;
      
    // iterate through points in the voxel
    auto* p = reinterpret_cast<const nanovdb::Vec3f*>(grid.blindData(0)) + leaf->minimum();
    begin = p + (offset == 0 ? 0 : leaf->getValue(offset - 1));
    end = p + leaf->getValue(offset);
    while (begin != end)
    {
      // convert from various internal voxel coordinate frames to world frame
      const nanovdb::Vec3f idx = nanovdb::Vec3f(ijk) + *begin++;
      const nanovdb::Vec3f pt = grid.indexToWorld(idx);

      // update minimum Z tracked so far
      min_z = std::isnan(min_z) ? pt[2] : fminf(min_z, pt[2]);
      // printf("Point (%f,%f,%f)", pt[0], pt[1], pt[2]);
    }
    
    // if we've found a point then break early; no points above this voxel will be smaller, axiomatically
    if (!std::isnan(min_z))
      break;
  }

  // update array element representing the minimum Z value in a column
  if (!std::isnan(min_z))
    deviceMap[j + i * (xmax - xmin + 1)] = min_z;
}

extern "C" void launch_ground_plane_kernel(const nanovdb::GridHandle<nanovdb::CudaDeviceBuffer>& gridHandle,
                                           const openvdb::CoordBBox& bbox,
                                           float* deviceMap,
                                           hipStream_t stream)
{
  // get a (raw) pointer to a NanoVDB grid of value type float on the GPU (uint32_t for PointDataGrid)
  auto* deviceGrid = gridHandle.deviceGrid<uint32_t>();

  // sanity check
  if (!deviceGrid || !deviceMap)
    throw std::runtime_error("Failed to load grid and / or mat to the GPU.");

  // set up GPU block / thread configuration
  auto round = [](int a, int b) { return (a + b - 1) / b; };
  constexpr dim3 threadsPerBlock(8, 8);
  const dim3 numBlocks(round(bbox.dim().x(), threadsPerBlock.x), round(bbox.dim().y(), threadsPerBlock.y));

  // kernel syntax:  <<<blocks per grid, threads per block, dynamic shared memory per block, stream >>>
  min_z_kernel<<<numBlocks, threadsPerBlock, 0, stream>>>(
    *deviceGrid, deviceMap,
    bbox.min().x(), bbox.max().x(),
    bbox.min().y(), bbox.max().y(),
    bbox.min().z(), bbox.max().z()
  );
}

} // namespace ovm::ops