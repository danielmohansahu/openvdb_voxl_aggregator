/* Simple ground plane extraction operation.
 *
 * Ground plane extraction operations.
 */

// STL
#include <iostream>

// NanoVDB
#include <nanovdb/NanoVDB.h>

// OVM
#include <openvdb_voxel_mapper/types.h>

namespace ovm::ops
{

extern "C" void launch_ground_plane_kernel(const nanovdb::NanoGrid<uint32_t>*,
                                           const nanovdb::NanoGrid<uint32_t>*,
                                           hipStream_t stream,
                                           ovm::Map::MapT& result)
{
  // @TODO!
}

} // namespace ovm::ops